#include "cudaArray_wrapper.h"




CudaArray_wrapper::CudaArray_wrapper(int width, int height, int depth)
{
	_width = width;
	_height = height;
	_depth = depth;

// Allocate depthmaps array.
	hipChannelFormatDesc fmt = hipCreateChannelDesc<char>();	
	struct hipExtent extent = make_hipExtent(_width, _height, _depth);	
	CUDA_SAFE_CALL(hipMalloc3DArray(&_array3D,&fmt,extent, hipArrayLayered));	

}

void CudaArray_wrapper::array3DCopy(char *img,  enum hipMemcpyKind kind)	// It must be one of hipMemcpyHostToHost, hipMemcpyHostToDevice, hipMemcpyDeviceToHost, or hipMemcpyDeviceToDevice
{
	if(kind == hipMemcpyHostToDevice)
	{
		struct hipExtent extent = make_hipExtent(_width, _height, _depth); //If a CUDA array is participating in the copy, the extent is defined in terms of that array's elements	
		//copy data
		struct hipMemcpy3DParms params = {0};
		params.extent = extent;
		params.kind = /*hipMemcpyHostToDevice*/ kind; 
		params.dstArray = _array3D;
		params.srcPtr = make_hipPitchedPtr((void*)img,_width*sizeof(char),_width,_height);
		CUDA_SAFE_CALL(hipMemcpy3D(&params));
		//hipChannelFormatDesc fmt1 = hipCreateChannelDesc<float>();
		//CUDA_SAFE_CALL(hipBindTextureToArray(depthmapsTex,_array3D,fmt));	// bind to texture
		//depthmapsTex.normalized = true;
	}
	else if(kind == hipMemcpyDeviceToHost)
	{ 
		struct hipMemcpy3DParms params = {0};
		params.extent = make_hipExtent(_width, _height, _depth);
		params.kind = kind; 
		params.dstPtr = make_hipPitchedPtr((void*)img,_width*sizeof(char),_width,_height);
		params.srcArray = _array3D;
		CUDA_SAFE_CALL(hipMemcpy3D(&params));
	}
}